#include "hip/hip_runtime.h"
#include "../../include/CUDA/CannyCUDA.cuh"

cv::Mat cuda_canny_input_image, cuda_canny_output_image;
cv::String cuda_canny_input_file, cuda_canny_output_file;

StopWatchInterface *canny_cuda_timer = nullptr;

hipStream_t stream;
uint8_t *input_pixels = nullptr, *output_pixels = nullptr, *segment_pixels = nullptr, *final_result = nullptr;
double *gradient_pixels = nullptr, *max_pixels = nullptr, *gaussian_kernel_gpu = nullptr;
int8_t* sobel_kernel_x_gpu = nullptr, *sobel_kernel_y_gpu = nullptr;

__global__ void gaussianBlur(const uint8_t* input_data, uint8_t* output_data, int image_width, int image_height, double* gaussian_kernel)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	const int offset_xy = ((KERNEL_SIZE - 1) / 2);

	if ((id >= 0 && id < image_height * image_width))
    {
        double kernelSum = 0;
        double blurPixel = 0;

        for (int i = 0; i < KERNEL_SIZE; i++) 
        {
            for (int j = 0; j < KERNEL_SIZE; j++) 
            {
                if (((id + ((i - offset_xy) * image_width) + j - offset_xy) >= 0) && 
                    ((id + ((i - offset_xy) * image_width) + j - offset_xy) <= image_height * image_width - 1) && 
                    (((id % image_width) + j - offset_xy) >= 0) && 
                    (((id % image_width) + j - offset_xy) <= (image_width - 1))) 
                    {
                        blurPixel = blurPixel + gaussian_kernel[i * KERNEL_SIZE + j] * input_data[id + ((i - offset_xy) * image_width) + j - offset_xy];
                        kernelSum = kernelSum + gaussian_kernel[i * KERNEL_SIZE + j];
                    }
            }
        }
        
        output_data[id] = (uint8_t)(blurPixel / kernelSum);
    }
}

__global__ void sobelFilter(double* gradient_pixels, uint8_t* segment_pixels, const uint8_t* input_data, int image_width, int image_height, int8_t* sobel_kernel_x, int8_t* sobel_kernel_y ) 
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	if ((id >= 0 && id < image_height * image_width))
    {
        int offset_xy = 1;
        double convolve_X = 0.0;
        double convolve_Y = 0.0;
        int k = 0;
        int segment = 0;

        int x = id % image_width;
        int y = id / image_width;
        
        if (x < offset_xy || x >= image_width - offset_xy || y < offset_xy || y >= image_height - offset_xy)
            return;
        
        int src_id = x + (y * image_width);

        for (int ky = -offset_xy; ky <= offset_xy; ky++) {
            for (int kx = -offset_xy; kx <= offset_xy; kx++) {
                convolve_X += input_data[src_id + (kx + (ky * image_width))] * sobel_kernel_x[k];
                convolve_Y += input_data[src_id + (kx + (ky * image_width))] * sobel_kernel_y[k];
                k++;
            }
        }

        if (convolve_X == 0.0 || convolve_Y == 0.0) 
        {
            gradient_pixels[src_id] = 0;
        }
        else 
        {
            gradient_pixels[src_id] = ((std::sqrt((convolve_X * convolve_X) + (convolve_Y * convolve_Y))));
            double theta = std::atan2(convolve_Y, convolve_X);
            theta = theta * (360.0 / (2.0 * M_PI));

            if ((theta <= 22.5 && theta >= -22.5) || (theta <= -157.5) || (theta >= 157.5))
                segment = 1;
            else if ((theta > 22.5 && theta <= 67.5) || (theta > -157.5 && theta <= -112.5))
                segment = 2;
            else if ((theta > 67.5 && theta <= 112.5) || (theta >= -112.5 && theta < -67.5))
                segment = 3;
            else if ((theta >= -67.5 && theta < -22.5) || (theta > 112.5 && theta < 157.5))
                segment = 4;
        }

        segment_pixels[src_id] = (uint8_t)segment;
    }
}

__global__ void nonMaxSuppression(double* max_pixels, double* gradient_pixels, uint8_t* segment_pixels, int image_width, int image_height) 
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	if ((id >= 0 && id < image_height * image_width))
    {
        switch (segment_pixels[id]) 
        {
            case 1:
                if (segment_pixels[id - 1] >= gradient_pixels[id] || gradient_pixels[id + 1] > gradient_pixels[id])
                    max_pixels[id] = 0;
            break;

            case 2:
                if (gradient_pixels[id - (image_width - 1)] >= gradient_pixels[id] || gradient_pixels[id + (image_width - 1)] > gradient_pixels[id])
                    max_pixels[id] = 0;
            break;

            case 3:
                if (gradient_pixels[id - (image_width)] >= gradient_pixels[id] || gradient_pixels[id + (image_width)] > gradient_pixels[id])
                    max_pixels[id] = 0;
            break;

            case 4:
                if (gradient_pixels[id - (image_width + 1)] >= gradient_pixels[id] || gradient_pixels[id + (image_width + 1)] > gradient_pixels[id])
                    max_pixels[id] = 0;
            break;

            default:
                max_pixels[id] = 0;
            break;
        }
    }
}

__global__ void doubleThreshold(uint8_t* out, double* max_pixels, int strong_threshold, int weak_threshold, int image_width, int image_height) 
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	if ((id >= 0 && id < image_height * image_width))
    {
        if (max_pixels[id] > strong_threshold)
		    out[id] = 255;
        else if (max_pixels[id] > weak_threshold)
            out[id] = 100;
        else
            out[id] = 0;
    }
}

__global__ void edgeHysteresis(uint8_t* out, uint8_t* in, int image_width, int image_height) 
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	if ((id >= 0 && id < image_height * image_width))
    {
        if (in[id] == 100) 
        {
            if (in[id - 1] == 255 || in[id + 1] == 255 ||
                in[id - image_width] == 255 || in[id + image_width] == 255 ||
                in[id - image_width - 1] == 255 || in[id - image_width + 1] == 255 ||
                in[id + image_width - 1] == 255 || in[id + image_width + 1] == 255)
                {
                    out[id] = 255;
                }
                
            else
                out[id] = 0;
        }
	}
}

void cuda_canny_mem_alloc(void** dev_ptr, size_t size)
{
    hipError_t result = hipMalloc(dev_ptr, size);
    if (result != hipSuccess)
    {
        std::cerr << std::endl << "Failed to allocate memory to " << dev_ptr << " : " << hipGetErrorString(result) << " ... Exiting !!!" << std::endl;
        canny_cuda_cleanup();
        exit(EXIT_FAILURE);
    }
}

void cuda_canny_mem_copy(void *dst, const void *src, size_t count, hipMemcpyKind kind)
{
    hipError_t result = hipMemcpy(dst, src, count, kind);
    if (result != hipSuccess)
    {
        std::cerr << std::endl << "Failed to copy memory from " << src << " to " << dst << " : " << hipGetErrorString(result) << " ... Exiting !!!" << std::endl;
        canny_cuda_cleanup();
        exit(EXIT_FAILURE);
    }
}

void cuda_canny_mem_free(void* dev_ptr)
{
    if (dev_ptr)
    {
        hipFree(dev_ptr);
        dev_ptr = nullptr;
    }
}

void run_canny_operator(uint8_t *input_image_data, uint8_t *output_image_data, int image_width, int image_height)
{
    // Variable Declarations
	const double gaussian_kernel[9] = 
    {
		1, 2, 1,
		2, 4, 2,
		1, 2, 1
	};

	const int8_t sobel_kernel_x[] = 
    {   
        -1, 0, 1,
		-2, 0, 2,
		-1, 0, 1 
    };

	const int8_t sobel_kernel_y[] = 
    {    
        1, 2, 1,
		0, 0, 0,
		-1,-2,-1 
    };

	const int NUM_BLOCKS = (image_height * image_width) / THREADS_PER_BLOCK;

	cuda_canny_mem_alloc((void**)&input_pixels, sizeof(uint8_t) * image_height * image_width);
	cuda_canny_mem_alloc((void**)&output_pixels, sizeof(uint8_t) * image_height * image_width);
	cuda_canny_mem_alloc((void**)&gradient_pixels, sizeof(double) * image_height * image_width);
	cuda_canny_mem_alloc((void**)&final_result, sizeof(uint8_t) * image_height * image_width);
	cuda_canny_mem_alloc((void**)&max_pixels, sizeof(double) * image_height * image_width);
	cuda_canny_mem_alloc((void**)&segment_pixels, sizeof(uint8_t) * image_height * image_width);
	cuda_canny_mem_alloc((void**)&gaussian_kernel_gpu, sizeof(double) * KERNEL_SIZE * KERNEL_SIZE);
	cuda_canny_mem_alloc((void**)&sobel_kernel_x_gpu, sizeof(int8_t) * 3 * 3);
	cuda_canny_mem_alloc((void**)&sobel_kernel_y_gpu, sizeof(int8_t) * 3 * 3);

	cuda_canny_mem_copy(input_pixels, input_image_data, image_height * image_width * sizeof(uint8_t), hipMemcpyHostToDevice);
	cuda_canny_mem_copy(gaussian_kernel_gpu, gaussian_kernel, sizeof(double) * KERNEL_SIZE * KERNEL_SIZE, hipMemcpyHostToDevice);
	cuda_canny_mem_copy(sobel_kernel_x_gpu, sobel_kernel_x, sizeof(int8_t) * KERNEL_SIZE * KERNEL_SIZE, hipMemcpyHostToDevice);
	cuda_canny_mem_copy(sobel_kernel_y_gpu, sobel_kernel_y, sizeof(int8_t) * KERNEL_SIZE * KERNEL_SIZE, hipMemcpyHostToDevice);

    sdkCreateTimer(&canny_cuda_timer);
	hipStreamCreate(&stream);

    sdkStartTimer(&canny_cuda_timer);
	gaussianBlur<<<NUM_BLOCKS, THREADS_PER_BLOCK, GRID, stream>>>(input_pixels, output_pixels, image_width, image_height, gaussian_kernel_gpu);
    sobelFilter<<<NUM_BLOCKS, THREADS_PER_BLOCK, GRID, stream>>>(gradient_pixels, segment_pixels, output_pixels, image_width, image_height, sobel_kernel_x_gpu, sobel_kernel_y_gpu);
    cuda_canny_mem_copy(max_pixels, gradient_pixels, image_height * image_width * sizeof(double), hipMemcpyDeviceToDevice);
	nonMaxSuppression<<<NUM_BLOCKS, THREADS_PER_BLOCK, GRID, stream >>>(max_pixels, gradient_pixels, segment_pixels, image_width, image_height);
	doubleThreshold<<<NUM_BLOCKS, THREADS_PER_BLOCK, GRID, stream>>>(output_pixels, max_pixels, CUDA_THRESHOLD * 3, CUDA_THRESHOLD, image_width, image_height);
	cuda_canny_mem_copy(final_result, output_pixels, image_height * image_width * sizeof(uint8_t), hipMemcpyDeviceToDevice);
	edgeHysteresis<<<NUM_BLOCKS, THREADS_PER_BLOCK, GRID, stream>>>(final_result, output_pixels, image_width, image_height);
    sdkStopTimer(&canny_cuda_timer);

	cuda_canny_mem_copy(output_image_data, final_result, image_width * image_height * sizeof(uint8_t), hipMemcpyDeviceToHost);
}

double canny_cuda(string input_file)
{
    cuda_canny_input_file = input_file;
    string output_file_name = filesystem::path(input_file).filename();
    cuda_canny_output_file = "/home/atharv/Downloads/Images/Output/Canny_CUDA_" + output_file_name;

    cuda_canny_input_image = cv::imread(cuda_canny_input_file, cv::IMREAD_GRAYSCALE);
    cuda_canny_output_image = cuda_canny_input_image.clone();

    run_canny_operator(cuda_canny_input_image.data, cuda_canny_output_image.data, cuda_canny_input_image.cols, cuda_canny_input_image.rows);

    double result = sdkGetTimerValue(&canny_cuda_timer);

    cuda_canny_output_image.convertTo(cuda_canny_output_image, CV_8UC1);

    cv::imwrite(cuda_canny_output_file, cuda_canny_output_image);

    canny_cuda_cleanup();

    return result;
}

void canny_cuda_cleanup(void)
{
	cuda_canny_mem_free(final_result);
    cuda_canny_mem_free(sobel_kernel_y_gpu);
    cuda_canny_mem_free(sobel_kernel_x_gpu);
    cuda_canny_mem_free(gaussian_kernel_gpu);
    cuda_canny_mem_free(segment_pixels);
    cuda_canny_mem_free(max_pixels);
    cuda_canny_mem_free(gradient_pixels);
    cuda_canny_mem_free(output_pixels);
    cuda_canny_mem_free(input_pixels);

    if (stream)
    {
        hipStreamDestroy(stream);
    }

    if (canny_cuda_timer)
    {
        sdkDeleteTimer(&canny_cuda_timer);
        canny_cuda_timer = nullptr;
    }

    cuda_canny_output_image.release();
    cuda_canny_input_image.release();
}
