#include "hip/hip_runtime.h"
#include "../../include/CUDA/CannyCUDA.cuh"

// Global Variables
uchar_t *input_pixels = nullptr, *output_pixels = nullptr, *segment_pixels = nullptr, *final_result = nullptr;
double *gradient_pixels = nullptr, *max_pixels = nullptr, *gaussian_kernel_gpu = nullptr;
schar_t* sobel_kernel_x_gpu = nullptr, *sobel_kernel_y_gpu = nullptr;

// CUDA Kernels
__global__ void gaussianBlur(const uchar_t* input_data, uchar_t* output_data, int image_width, int image_height, double* gaussian_kernel)
{
    // Code
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	const int offset_xy = ((CV_GAUSSIAN_KERNEL_SIZE - 1) / 2);

	if ((id >= 0 && id < image_height * image_width))
    {
        double kernelSum = 0;
        double blurPixel = 0;

        for (int i = 0; i < CV_GAUSSIAN_KERNEL_SIZE; i++) 
        {
            for (int j = 0; j < CV_GAUSSIAN_KERNEL_SIZE; j++) 
            {
                if (((id + ((i - offset_xy) * image_width) + j - offset_xy) >= 0) && 
                    ((id + ((i - offset_xy) * image_width) + j - offset_xy) <= image_height * image_width - 1) && 
                    (((id % image_width) + j - offset_xy) >= 0) && 
                    (((id % image_width) + j - offset_xy) <= (image_width - 1))) 
                    {
                        blurPixel = blurPixel + gaussian_kernel[i * CV_GAUSSIAN_KERNEL_SIZE + j] * input_data[id + ((i - offset_xy) * image_width) + j - offset_xy];
                        kernelSum = kernelSum + gaussian_kernel[i * CV_GAUSSIAN_KERNEL_SIZE + j];
                    }
            }
        }
        
        output_data[id] = (uchar_t)(blurPixel / kernelSum);
    }
}

__global__ void sobelFilter(double* gradient_pixels, uchar_t* segment_pixels, const uchar_t* input_data, int image_width, int image_height, schar_t* sobel_kernel_x, schar_t* sobel_kernel_y ) 
{
    // Code
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	if ((id >= 0 && id < image_height * image_width))
    {
        int offset_xy = 1;
        double convolve_X = 0.0;
        double convolve_Y = 0.0;
        int k = 0;
        int segment = 0;

        int x = id % image_width;
        int y = id / image_width;
        
        if (x < offset_xy || x >= image_width - offset_xy || y < offset_xy || y >= image_height - offset_xy)
            return;
        
        int src_id = x + (y * image_width);

        for (int ky = -offset_xy; ky <= offset_xy; ky++) {
            for (int kx = -offset_xy; kx <= offset_xy; kx++) {
                convolve_X += input_data[src_id + (kx + (ky * image_width))] * sobel_kernel_x[k];
                convolve_Y += input_data[src_id + (kx + (ky * image_width))] * sobel_kernel_y[k];
                k++;
            }
        }

        if (convolve_X == 0.0 || convolve_Y == 0.0) 
        {
            gradient_pixels[src_id] = 0;
        }
        else 
        {
            gradient_pixels[src_id] = ((std::sqrt((convolve_X * convolve_X) + (convolve_Y * convolve_Y))));
            double theta = std::atan2(convolve_Y, convolve_X);
            theta = theta * (360.0 / (2.0 * M_PI));

            if ((theta <= 22.5 && theta >= -22.5) || (theta <= -157.5) || (theta >= 157.5))
                segment = 1;
            else if ((theta > 22.5 && theta <= 67.5) || (theta > -157.5 && theta <= -112.5))
                segment = 2;
            else if ((theta > 67.5 && theta <= 112.5) || (theta >= -112.5 && theta < -67.5))
                segment = 3;
            else if ((theta >= -67.5 && theta < -22.5) || (theta > 112.5 && theta < 157.5))
                segment = 4;
        }

        segment_pixels[src_id] = (uchar_t)segment;
    }
}

__global__ void nonMaxSuppression(double* max_pixels, double* gradient_pixels, uchar_t* segment_pixels, int image_width, int image_height) 
{
    // Code
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	if ((id >= 0 && id < image_height * image_width))
    {
        switch (segment_pixels[id]) 
        {
            case 1:
                if (segment_pixels[id - 1] >= gradient_pixels[id] || gradient_pixels[id + 1] > gradient_pixels[id])
                    max_pixels[id] = 0;
            break;

            case 2:
                if (gradient_pixels[id - (image_width - 1)] >= gradient_pixels[id] || gradient_pixels[id + (image_width - 1)] > gradient_pixels[id])
                    max_pixels[id] = 0;
            break;

            case 3:
                if (gradient_pixels[id - (image_width)] >= gradient_pixels[id] || gradient_pixels[id + (image_width)] > gradient_pixels[id])
                    max_pixels[id] = 0;
            break;

            case 4:
                if (gradient_pixels[id - (image_width + 1)] >= gradient_pixels[id] || gradient_pixels[id + (image_width + 1)] > gradient_pixels[id])
                    max_pixels[id] = 0;
            break;

            default:
                max_pixels[id] = 0;
            break;
        }
    }
}

__global__ void doubleThreshold(uchar_t* out, double* max_pixels, int strong_threshold, int weak_threshold, int image_width, int image_height) 
{
    // Code
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	if ((id >= 0 && id < image_height * image_width))
    {
        if (max_pixels[id] > strong_threshold)
		    out[id] = 255;
        else if (max_pixels[id] > weak_threshold)
            out[id] = 100;
        else
            out[id] = 0;
    }
}

__global__ void edgeHysteresis(uchar_t* out, uchar_t* in, int image_width, int image_height) 
{
    // Code
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	if ((id >= 0 && id < image_height * image_width))
    {
        if (in[id] == 100) 
        {
            if (in[id - 1] == 255 || in[id + 1] == 255 ||
                in[id - image_width] == 255 || in[id + image_width] == 255 ||
                in[id - image_width - 1] == 255 || in[id - image_width + 1] == 255 ||
                in[id + image_width - 1] == 255 || in[id + image_width + 1] == 255)
                {
                    out[id] = 255;
                }
                
            else
                out[id] = 0;
        }
	}
}

void cuda_canny_mem_alloc(void** dev_ptr, size_t size)
{
    // Code
    hipError_t result = hipMalloc(dev_ptr, size);
    if (result != hipSuccess)
    {
        std::cerr << std::endl << "Failed to allocate memory to " << dev_ptr << " : " << hipGetErrorString(result) << " ... Exiting !!!" << std::endl;
        canny_cuda_cleanup();
        exit(EXIT_FAILURE);
    }
}

void cuda_canny_mem_copy(void *dst, const void *src, size_t count, hipMemcpyKind kind)
{
    // Code 
    hipError_t result = hipMemcpy(dst, src, count, kind);
    if (result != hipSuccess)
    {
        std::cerr << std::endl << "Failed to copy memory from " << src << " to " << dst << " : " << hipGetErrorString(result) << " ... Exiting !!!" << std::endl;
        canny_cuda_cleanup();
        exit(EXIT_FAILURE);
    }
}

void cuda_canny_mem_free(void** dev_ptr)
{
    // Code
    if (*dev_ptr)
    {
        hipFree(*dev_ptr);
        *dev_ptr = nullptr;
    }
}

double canny_operator(uchar_t *input_image_data, uchar_t *output_image_data, int image_width, int image_height)
{
    // Variable Declarations
    StopWatchInterface *canny_cuda_timer = nullptr;

	const double gaussian_kernel[9] = 
    {
		1, 2, 1,
		2, 4, 2,
		1, 2, 1
	};

	const schar_t sobel_kernel_x[] = 
    {   
        -1, 0, 1,
		-2, 0, 2,
		-1, 0, 1 
    };

	const schar_t sobel_kernel_y[] = 
    {    
        1, 2, 1,
		0, 0, 0,
		-1,-2,-1 
    };

	const int NUM_BLOCKS = (image_height * image_width) / THREADS_PER_BLOCK;

	cuda_canny_mem_alloc((void**)&input_pixels, sizeof(uchar_t) * image_height * image_width);
	cuda_canny_mem_alloc((void**)&output_pixels, sizeof(uchar_t) * image_height * image_width);
	cuda_canny_mem_alloc((void**)&gradient_pixels, sizeof(double) * image_height * image_width);
	cuda_canny_mem_alloc((void**)&final_result, sizeof(uchar_t) * image_height * image_width);
	cuda_canny_mem_alloc((void**)&max_pixels, sizeof(double) * image_height * image_width);
	cuda_canny_mem_alloc((void**)&segment_pixels, sizeof(uchar_t) * image_height * image_width);
	cuda_canny_mem_alloc((void**)&gaussian_kernel_gpu, sizeof(double) * CV_GAUSSIAN_KERNEL_SIZE * CV_GAUSSIAN_KERNEL_SIZE);
	cuda_canny_mem_alloc((void**)&sobel_kernel_x_gpu, sizeof(schar_t) * 3 * 3);
	cuda_canny_mem_alloc((void**)&sobel_kernel_y_gpu, sizeof(schar_t) * 3 * 3);

	cuda_canny_mem_copy(input_pixels, input_image_data, image_height * image_width * sizeof(uchar_t), hipMemcpyHostToDevice);
	cuda_canny_mem_copy(gaussian_kernel_gpu, gaussian_kernel, sizeof(double) * CV_GAUSSIAN_KERNEL_SIZE * CV_GAUSSIAN_KERNEL_SIZE, hipMemcpyHostToDevice);
	cuda_canny_mem_copy(sobel_kernel_x_gpu, sobel_kernel_x, sizeof(schar_t) * CV_GAUSSIAN_KERNEL_SIZE * CV_GAUSSIAN_KERNEL_SIZE, hipMemcpyHostToDevice);
	cuda_canny_mem_copy(sobel_kernel_y_gpu, sobel_kernel_y, sizeof(schar_t) * CV_GAUSSIAN_KERNEL_SIZE * CV_GAUSSIAN_KERNEL_SIZE, hipMemcpyHostToDevice);

    // CUDA Kernel Call
    sdkCreateTimer(&canny_cuda_timer);
    sdkStartTimer(&canny_cuda_timer);
    {
        gaussianBlur<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>(input_pixels, output_pixels, image_width, image_height, gaussian_kernel_gpu);
        sobelFilter<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>(gradient_pixels, segment_pixels, output_pixels, image_width, image_height, sobel_kernel_x_gpu, sobel_kernel_y_gpu);
        
        cuda_canny_mem_copy(max_pixels, gradient_pixels, image_height * image_width * sizeof(double), hipMemcpyDeviceToDevice);
        
        nonMaxSuppression<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>(max_pixels, gradient_pixels, segment_pixels, image_width, image_height);
        doubleThreshold<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>(output_pixels, max_pixels, CUDA_THRESHOLD * 3, CUDA_THRESHOLD, image_width, image_height);
        
        cuda_canny_mem_copy(final_result, output_pixels, image_height * image_width * sizeof(uchar_t), hipMemcpyDeviceToDevice);
        
        edgeHysteresis<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>(final_result, output_pixels, image_width, image_height);
    }
    sdkStopTimer(&canny_cuda_timer);

    // Get Execution Time
    double canny_time = sdkGetTimerValue(&canny_cuda_timer);
    sdkDeleteTimer(&canny_cuda_timer);
    canny_cuda_timer = nullptr;

	cuda_canny_mem_copy(output_image_data, final_result, image_width * image_height * sizeof(uchar_t), hipMemcpyDeviceToHost);

    return canny_time;
}

double canny_cuda(string input_file, string output_file)
{
    // Variable Declarations
    cv::Mat cuda_canny_input_image, cuda_canny_output_image;
    cv::String cuda_canny_input_file, cuda_canny_output_file;

    // Code
    cuda_canny_input_file = input_file;
    filesystem::path output_path = filesystem::path(input_file).filename();
    string output_file_name = output_path.string();

    #if (OS == 1)
        cuda_canny_output_file = output_file + "\\Canny_CUDA_" + output_file_name;
    #elif (OS == 2)
        cuda_canny_output_file = output_file + "/Canny_CUDA_" + output_file_name;
    #endif

    cuda_canny_input_image = cv::imread(cuda_canny_input_file, cv::IMREAD_GRAYSCALE);
    cuda_canny_output_image = cuda_canny_input_image.clone();

    double result = canny_operator(cuda_canny_input_image.data, cuda_canny_output_image.data, cuda_canny_input_image.cols, cuda_canny_input_image.rows);

    cuda_canny_output_image.convertTo(cuda_canny_output_image, CV_8UC1);

    cv::imwrite(cuda_canny_output_file, cuda_canny_output_image);

    canny_cuda_cleanup();
    cuda_canny_output_image.release();
    cuda_canny_input_image.release();

    return result;
}

void canny_cuda_cleanup(void)
{
    // Code
	cuda_canny_mem_free((void**)&final_result);
    cuda_canny_mem_free((void**)&sobel_kernel_y_gpu);
    cuda_canny_mem_free((void**)&sobel_kernel_x_gpu);
    cuda_canny_mem_free((void**)&gaussian_kernel_gpu);
    cuda_canny_mem_free((void**)&segment_pixels);
    cuda_canny_mem_free((void**)&max_pixels);
    cuda_canny_mem_free((void**)&gradient_pixels);
    cuda_canny_mem_free((void**)&output_pixels);
    cuda_canny_mem_free((void**)&input_pixels);
}
