#include "hip/hip_runtime.h"
#include "../../include/CUDA/SobelCUDA.cuh"

// Global Variables
unsigned char *device_input = nullptr, *device_output = nullptr;
float *host_kernel = nullptr, *device_kernel = nullptr;

__global__ void gaussianBlurKernel(unsigned char *cuda_sobel_input_image, unsigned char *cuda_sobel_output_image, int width, int height, float *kernel)
{
    // Code
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height)
    {
        float blur_pixel = 0.0f;
        int kernel_radius = CUDA_GAUSSIAN_KERNEL_SIZE / 2;

        for (int i = -kernel_radius; i <= kernel_radius; i++)
        {
            for (int j = -kernel_radius; j <= kernel_radius; j++)
            {
                int x_offset = x + i;
                int y_offset = y + j;

                if (x_offset >= 0 && x_offset < width && y_offset >= 0 && y_offset < height)
                {
                    int input_index = y_offset * width + x_offset;
                    int kernel_index = (i + kernel_radius) * CUDA_GAUSSIAN_KERNEL_SIZE + (j + kernel_radius);
                    blur_pixel = blur_pixel + static_cast<float>(cuda_sobel_input_image[input_index]) * kernel[kernel_index];
                }
            }
        }

        cuda_sobel_output_image[y * width + x] = static_cast<unsigned char>(blur_pixel);
    }
}

__global__ void sobelFilterKernel(unsigned char *cuda_sobel_input_image, unsigned char *cuda_sobel_output_image, unsigned int image_width, unsigned int image_height)
{
    // Variable Declarations
    int sobel_x[CUDA_SOBEL_KERNEL_SIZE][CUDA_SOBEL_KERNEL_SIZE] = 
    {
        { -1, 0, 1 },
        { -2, 0, 2 },
        { -1, 0, 1 }
    };

    int sobel_y[CUDA_SOBEL_KERNEL_SIZE][CUDA_SOBEL_KERNEL_SIZE] = 
    {
        { -1, -2, -1 },
        {  0,  0,  0 },
        {  1,  2,  1 }
    };

    // Code
    int num_rows = blockIdx.x * blockDim.x + threadIdx.x;
    int num_columns = blockIdx.y * blockDim.y + threadIdx.y;

    int index = (num_rows * image_width) + num_columns;

    if ((num_columns < (image_width - 1)) && (num_rows < (image_height - 1)))
    {
        float gradient_x =  (cuda_sobel_input_image[index] * sobel_x[0][0]) + (cuda_sobel_input_image[index + 1] * sobel_x[0][1]) + (cuda_sobel_input_image[index + 2] * sobel_x[0][2]) +
                            (cuda_sobel_input_image[index] * sobel_x[1][0]) + (cuda_sobel_input_image[index + 1] * sobel_x[1][1]) + (cuda_sobel_input_image[index + 2] * sobel_x[1][2]) +
                            (cuda_sobel_input_image[index] * sobel_x[2][0]) + (cuda_sobel_input_image[index + 1] * sobel_x[2][1]) + (cuda_sobel_input_image[index + 2] * sobel_x[2][2]);

        float gradient_y =  (cuda_sobel_input_image[index] * sobel_y[0][0]) + (cuda_sobel_input_image[index + 1] * sobel_y[0][1]) + (cuda_sobel_input_image[index + 2] * sobel_y[0][2]) +
                            (cuda_sobel_input_image[index] * sobel_y[1][0]) + (cuda_sobel_input_image[index + 1] * sobel_y[1][1]) + (cuda_sobel_input_image[index + 2] * sobel_y[1][2]) +
                            (cuda_sobel_input_image[index] * sobel_y[2][0]) + (cuda_sobel_input_image[index + 1] * sobel_y[2][1]) + (cuda_sobel_input_image[index + 2] * sobel_y[2][2]);

        float gradient = sqrtf(gradient_x * gradient_x + gradient_y * gradient_y);

        if (gradient > 255)
            gradient = 255;

        if (gradient < 0)
            gradient = 0;

        __syncthreads();

        cuda_sobel_output_image[index] = gradient;
    }
}

void cuda_sobel_mem_alloc(void** dev_ptr, size_t size)
{
    // Code
    hipError_t result = hipMalloc(dev_ptr, size);
    if (result != hipSuccess)
    {
        std::cerr << std::endl << "Failed to allocate memory to " << dev_ptr << " : " << hipGetErrorString(result) << " ... Exiting !!!" << std::endl;
        sobel_cuda_cleanup();
        exit(EXIT_FAILURE);
    }
}

void cuda_sobel_mem_copy(void *dst, const void *src, size_t count, hipMemcpyKind kind)
{
    // Code
    hipError_t result = hipMemcpy(dst, src, count, kind);
    if (result != hipSuccess)
    {
        std::cerr << std::endl << "Failed to copy memory from " << src << " to " << dst << " : " << hipGetErrorString(result) << " ... Exiting !!!" << std::endl;
        sobel_cuda_cleanup();
        exit(EXIT_FAILURE);
    }
}

void cuda_sobel_mem_free(void** dev_ptr)
{
    // Code
    if (*dev_ptr)
    {
        hipFree(*dev_ptr);
        *dev_ptr = nullptr;
    }
}

double sobel_operator(cv::Mat *input_image, cv::Mat *output_image)
{
    // Variable Declarations
    float kernel_sum = 0.0f;
    float sigma = 1.0f;
    StopWatchInterface *sobel_cuda_timer = nullptr;

    // Code

    // Get Image Properties
    int image_width = input_image->cols;
    int image_height = input_image->rows;
    int image_size = image_height * image_width * sizeof(unsigned char);

    // Create Gaussian Kernel
    host_kernel = new float[CUDA_GAUSSIAN_KERNEL_SIZE * CUDA_GAUSSIAN_KERNEL_SIZE];
    int kernel_radius = CUDA_GAUSSIAN_KERNEL_SIZE / 2;

    for (int i = -kernel_radius; i <= kernel_radius; i++) 
    {
        for (int j = -kernel_radius; j <= kernel_radius; j++)
        {
            int index = (i + kernel_radius) * kernel_radius + (j + kernel_radius);
            host_kernel[index] = exp(-(i * i + j + j) / (2.0f * sigma * sigma));
            kernel_sum = kernel_sum + host_kernel[index];
        }
    }

    for (int i = 0; i < CUDA_GAUSSIAN_KERNEL_SIZE * CUDA_GAUSSIAN_KERNEL_SIZE; i++)
        host_kernel[i] = host_kernel[i] / kernel_sum;

    cuda_sobel_mem_alloc((void **)&device_input, image_size);
    cuda_sobel_mem_alloc((void **)&device_output, image_size);
    cuda_sobel_mem_alloc((void **)&device_kernel, CUDA_GAUSSIAN_KERNEL_SIZE * CUDA_GAUSSIAN_KERNEL_SIZE * sizeof(float));
    
    cuda_sobel_mem_copy(device_input, input_image->data, image_size, hipMemcpyHostToDevice);
    cuda_sobel_mem_copy(device_kernel, host_kernel, CUDA_GAUSSIAN_KERNEL_SIZE * CUDA_GAUSSIAN_KERNEL_SIZE * sizeof(float), hipMemcpyHostToDevice);

    // Kernel Configuration
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(image_height, image_width);

    // CUDA Kernel
    sdkCreateTimer(&sobel_cuda_timer);
    sdkStartTimer(&sobel_cuda_timer);
    {
        gaussianBlurKernel<<<dimGrid, dimBlock>>>(device_input, device_output, image_width, image_height, device_kernel);
        sobelFilterKernel<<<dimGrid, dimBlock>>>(device_input, device_output, input_image->cols, input_image->rows);
    }
    sdkStopTimer(&sobel_cuda_timer);

    // Get Execution Time
    double sobel_time = sdkGetTimerValue(&sobel_cuda_timer);
    sdkDeleteTimer(&sobel_cuda_timer);
    sobel_cuda_timer = nullptr;

    cuda_sobel_mem_copy(output_image->data, device_output, image_size, hipMemcpyDeviceToHost);

    return sobel_time;
}

double sobel_cuda(string input_file, string output_file)
{
    // Variable Declarations
    cv::String cuda_sobel_input_file, cuda_sobel_output_file;
    cv::Mat cuda_sobel_input_image, cuda_sobel_output_image;

    // Code

    // Input and Output File
    cuda_sobel_input_file = input_file;
    filesystem::path output_path = filesystem::path(input_file).filename();
    string output_file_name = output_path.string();

    #if (OS == 1)
        cuda_sobel_output_file = output_file + "\\Sobel_CUDA_" + output_file_name;
    #elif (OS == 2)
        cuda_sobel_output_file = output_file + "/Sobel_CUDA_" + output_file_name;
    #endif

    // Reading Input Image
    cuda_sobel_input_image = cv::imread(cuda_sobel_input_file, cv::IMREAD_GRAYSCALE);
    cuda_sobel_output_image = cuda_sobel_input_image.clone();

    double result = sobel_operator(&cuda_sobel_input_image, &cuda_sobel_output_image);

    cuda_sobel_output_image.convertTo(cuda_sobel_output_image, CV_8UC1);

    cv::imwrite(cuda_sobel_output_file, cuda_sobel_output_image);

    // Cleanup Code
    sobel_cuda_cleanup();
    cuda_sobel_output_image.release();
    cuda_sobel_input_image.release();

    return result;
}

void sobel_cuda_cleanup(void)
{
    // Code
    cuda_sobel_mem_free((void**)&device_kernel);
    cuda_sobel_mem_free((void**)&device_output);
    cuda_sobel_mem_free((void**)&device_input);

    if (host_kernel)
    {
        delete[] host_kernel;
        host_kernel = nullptr;
    }
}
