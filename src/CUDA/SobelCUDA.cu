#include "hip/hip_runtime.h"
#include "../../include/CUDA/SobelCUDA.cuh"

cv::Mat cuda_sobel_input_image, cuda_sobel_output_image;
cv::String cuda_sobel_input_file, cuda_sobel_output_file;
StopWatchInterface *sobelCudaTimer = nullptr;
unsigned char *deviceInput = nullptr, *deviceOutput = nullptr;
float *hostKernel = nullptr, *deviceKernel = nullptr;

__global__ void gaussianBlurKernel(unsigned char *cuda_sobel_input_image, unsigned char *cuda_sobel_output_image, int width, int height, float *kernel)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height)
    {
        float blurPixel = 0.0f;
        int kernelRadius = GAUSSIAN_KERNEL_SIZE / 2;

        for (int i = -kernelRadius; i <= kernelRadius; i++)
        {
            for (int j = -kernelRadius; j <= kernelRadius; j++)
            {
                int xOffset = x + i;
                int yOffset = y + j;

                if (xOffset >= 0 && xOffset < width && yOffset >= 0 && yOffset < height)
                {
                    int inputIndex = yOffset * width + xOffset;
                    int kernelIndex = (i + kernelRadius) * GAUSSIAN_KERNEL_SIZE + (j + kernelRadius);
                    blurPixel = blurPixel + static_cast<float>(cuda_sobel_input_image[inputIndex]) * kernel[kernelIndex];
                }
            }
        }

        cuda_sobel_output_image[y * width + x] = static_cast<unsigned char>(blurPixel);
    }
}

__global__ void sobelFilterKernel(unsigned char *cuda_sobel_input_image, unsigned char *cuda_sobel_output_image, unsigned int image_width, unsigned int image_height)
{
    int sobel_x[SOBEL_KERNEL_SIZE][SOBEL_KERNEL_SIZE] = {
        { -1, 0, 1 },
        { -2, 0, 2 },
        { -1, 0, 1 }
    };

    int sobel_y[SOBEL_KERNEL_SIZE][SOBEL_KERNEL_SIZE] = {
        { -1, -2, -1 },
        { 0, 0, 0 },
        { 1, 2, 1 }
    };

    int num_rows = blockIdx.x * blockDim.x + threadIdx.x;
    int num_columns = blockIdx.y * blockDim.y + threadIdx.y;

    int index = (num_rows * image_width) + num_columns;

    if ((num_columns < (image_width - 1)) && (num_rows < (image_height - 1)))
    {
        float gradient_x =  (cuda_sobel_input_image[index] * sobel_x[0][0]) + (cuda_sobel_input_image[index + 1] * sobel_x[0][1]) + (cuda_sobel_input_image[index + 2] * sobel_x[0][2]) +
                            (cuda_sobel_input_image[index] * sobel_x[1][0]) + (cuda_sobel_input_image[index + 1] * sobel_x[1][1]) + (cuda_sobel_input_image[index + 2] * sobel_x[1][2]) +
                            (cuda_sobel_input_image[index] * sobel_x[2][0]) + (cuda_sobel_input_image[index + 1] * sobel_x[2][1]) + (cuda_sobel_input_image[index + 2] * sobel_x[2][2]);

        float gradient_y =  (cuda_sobel_input_image[index] * sobel_y[0][0]) + (cuda_sobel_input_image[index + 1] * sobel_y[0][1]) + (cuda_sobel_input_image[index + 2] * sobel_y[0][2]) +
                            (cuda_sobel_input_image[index] * sobel_y[1][0]) + (cuda_sobel_input_image[index + 1] * sobel_y[1][1]) + (cuda_sobel_input_image[index + 2] * sobel_y[1][2]) +
                            (cuda_sobel_input_image[index] * sobel_y[2][0]) + (cuda_sobel_input_image[index + 1] * sobel_y[2][1]) + (cuda_sobel_input_image[index + 2] * sobel_y[2][2]);

        float gradient = sqrtf(gradient_x * gradient_x + gradient_y * gradient_y);

        if (gradient > 255)
            gradient = 255;

        if (gradient < 0)
            gradient = 0;

        __syncthreads();

        cuda_sobel_output_image[index] = gradient;
    }
}

void cuda_sobel_mem_alloc(void** devPtr, size_t size)
{
    hipError_t result = hipMalloc(devPtr, size);
    if (result != hipSuccess)
    {
        std::cerr << std::endl << "Failed to allocate memory to " << devPtr << " : " << hipGetErrorString(result) << " ... Exiting !!!" << std::endl;
        sobelCleanup();
        exit(EXIT_FAILURE);
    }
}

void cuda_sobel_mem_copy(void *dst, const void *src, size_t count, hipMemcpyKind kind)
{
    hipError_t result = hipMemcpy(dst, src, count, kind);
    if (result != hipSuccess)
    {
        std::cerr << std::endl << "Failed to copy memory from " << src << " to " << dst << " : " << hipGetErrorString(result) << " ... Exiting !!!" << std::endl;
        sobelCleanup();
        exit(EXIT_FAILURE);
    }
}

void cuda_sobel_mem_free(void* devPtr)
{
    if (devPtr)
    {
        hipFree(devPtr);
        devPtr = nullptr;
    }
}

void runSobelOperator(cv::Mat *inputImage, cv::Mat *outputImage)
{
    // Variable Declarations
    hipError_t result;
    float kernelSum = 0.0f;
    float sigma = 1.0f;

    int imageWidth = inputImage->cols;
    int imageHeight = inputImage->rows;
    int imageSize = imageHeight * imageWidth * sizeof(unsigned char);
    
    // Create Gaussian Kernel
    hostKernel = new float[GAUSSIAN_KERNEL_SIZE * GAUSSIAN_KERNEL_SIZE];
    int kernelRadius = GAUSSIAN_KERNEL_SIZE / 2;

    for (int i = -kernelRadius; i <= kernelRadius; i++) 
    {
        for (int j = -kernelRadius; j <= kernelRadius; j++)
        {
            int index = (i + kernelRadius) * kernelRadius + (j + kernelRadius);
            hostKernel[index] = exp(-(i * i + j + j) / (2.0f * sigma * sigma));
            kernelSum = kernelSum + hostKernel[index];
        }
    }

    for (int i = 0; i < GAUSSIAN_KERNEL_SIZE * GAUSSIAN_KERNEL_SIZE; i++)
    {
        hostKernel[i] = hostKernel[i] / kernelSum;
    }

    sdkCreateTimer(&sobelCudaTimer);

    result = hipMalloc((void **)&deviceInput, imageSize);
    if (result != hipSuccess)
    {
        std::cerr << std::endl << "hipMalloc() Failed For Input Image ... Exiting !!!" << std::endl;
        exit(EXIT_FAILURE);
    }

    result = hipMalloc((void **)&deviceOutput, imageSize);
    if (result != hipSuccess)
    {
        std::cerr << std::endl << "hipMalloc() Failed For Output Image ... Exiting !!!" << std::endl;
        exit(EXIT_FAILURE);
    }

    result = hipMalloc((void **)&deviceKernel, GAUSSIAN_KERNEL_SIZE * GAUSSIAN_KERNEL_SIZE * sizeof(float));
    if (result != hipSuccess)
    {
        std::cerr << std::endl << "hipMalloc() Failed For Device Kernel ... Exiting !!!" << std::endl;
        exit(EXIT_FAILURE);
    }

    result = hipMemcpy(deviceInput, inputImage->data, imageSize, hipMemcpyHostToDevice);
    if (result != hipSuccess)
    {
        std::cerr << std::endl << "hipMemcpy() Failed For Input Image ... Exiting !!!" << std::endl;
        exit(EXIT_FAILURE);
    }

    result = hipMemcpy(deviceKernel, hostKernel, GAUSSIAN_KERNEL_SIZE * GAUSSIAN_KERNEL_SIZE * sizeof(float), hipMemcpyHostToDevice);
    if (result != hipSuccess)
    {
        std::cerr << std::endl << "hipMemcpy() Failed For Device Kernel ... Exiting !!!" << std::endl;
        exit(EXIT_FAILURE);
    }

    // Kernel Configuration
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(imageHeight, imageWidth);

    sdkStartTimer(&sobelCudaTimer);
    gaussianBlurKernel<<<dimGrid, dimBlock>>>(deviceInput, deviceOutput, imageWidth, imageHeight, deviceKernel);
    sobelFilterKernel<<<dimGrid, dimBlock>>>(deviceInput, deviceOutput, inputImage->cols, inputImage->rows);
    sdkStopTimer(&sobelCudaTimer);

    result = hipMemcpy(outputImage->data, deviceOutput, imageSize, hipMemcpyDeviceToHost);
    if (result != hipSuccess)
    {
        std::cerr << std::endl << "hipMemcpy() Failed For Output Image ... Exiting !!!" << std::endl;
        exit(EXIT_FAILURE);
    }
}

void sobelCUDA(int image_number)
{
    switch(image_number)
    {
        case 1:
            cuda_sobel_input_file = "Images\\Input\\img1.jpg";
            cuda_sobel_output_file = "Images\\Output\\Sobel-CUDA-1.jpg";
        break;
        case 2:
            cuda_sobel_input_file = "Images\\Input\\img2.jpg";
            cuda_sobel_output_file = "Images\\Output\\Sobel-CUDA-2.jpg";
        break;
        case 3:
            cuda_sobel_input_file = "Images\\Input\\img3.jpg";
            cuda_sobel_output_file = "Images\\Output\\Sobel-CUDA-3.jpg";
        break;
        case 4:
            cuda_sobel_input_file = "Images\\Input\\img4.jpg";
            cuda_sobel_output_file = "Images\\Output\\Sobel-CUDA-4.jpg";
        break;
        case 5:
            cuda_sobel_input_file = "Images\\Input\\img5.jpg";
            cuda_sobel_output_file = "Images\\Output\\Sobel-CUDA-5.jpg";
        break;
        default:
            std::cerr << std::endl << "Error ... Please Enter Valid Number ... Exiting !!!" << std::endl;
            sobelCleanup();
            exit(EXIT_FAILURE);
        break;
    }

    cuda_sobel_input_image = cv::imread(cuda_sobel_input_file, cv::IMREAD_GRAYSCALE);
    cuda_sobel_output_image = cuda_sobel_input_image.clone();

    runSobelOperator(&cuda_sobel_input_image, &cuda_sobel_output_image);

    std::cout << std::endl << "Time for Sobel Operator using CUDA (GPU) : " << sdkGetTimerValue(&sobelCudaTimer) << " ms" << std::endl;

    cuda_sobel_output_image.convertTo(cuda_sobel_output_image, CV_8UC1);

    cv::imwrite(cuda_sobel_output_file, cuda_sobel_output_image);

    sobelCleanup();
}

void sobelCleanup(void)
{
    if (deviceKernel)
    {
        hipFree(deviceKernel);
        deviceKernel = nullptr;
    }

    if (deviceOutput)
    {
        hipFree(deviceOutput);
        deviceOutput = nullptr;
    }

    if (deviceInput)
    {
        hipFree(deviceInput);
        deviceInput = nullptr;
    }

    if (hostKernel)
    {
        delete[] hostKernel;
        hostKernel = nullptr;
    }

    if (sobelCudaTimer)
    {
        sdkDeleteTimer(&sobelCudaTimer);
        sobelCudaTimer = nullptr;
    }

    cuda_sobel_output_image.release();
    cuda_sobel_input_image.release();
}
